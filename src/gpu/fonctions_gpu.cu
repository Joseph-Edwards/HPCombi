#include <stdio.h>
#include <stdint.h>
#include "kernels.cuh"

void shufl_gpu(const uint8_t* x, const uint8_t* y, uint8_t* z, const size_t Size)
{
	// Memory allocation on GPU
	uint8_t *d_x, *d_y;
	hipMalloc((void**)&d_x, Size*sizeof(uint8_t));
	hipMalloc((void**)&d_y, Size*sizeof(uint8_t));

	// Definition of grid and block sizes
	dim3 block(Size,1);
	dim3 grid((Size+block.x-1)/block.x,1);

	// Copy CPU to GPU
	hipMemcpy(d_x, x, Size*sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, Size*sizeof(uint8_t), hipMemcpyHostToDevice);
	
		// Computation
		permute_gpu<<<grid, block>>>(d_x, d_y, Size);
	
	//Copy GPU to CPU
	hipMemcpy(z, d_x, Size*sizeof(uint8_t), hipMemcpyDeviceToHost);
	
	// Free GPU memory
	hipFree(d_x);
	hipFree(d_y);
}
